#include "hip/hip_runtime.h"
#include "common.h"

// density of the inverse gaussian distribution
__device__ double dinvgauss(double x, double mu, double lambda)
{
    // TODO would be nice to assert that x > 0 and lambda = 0

    double x_minus_mu = x - mu;
    return sqrt(lambda / (2 * HIP_PI * pow(x, 3.0))) * exp((-lambda * x_minus_mu * x_minus_mu) / (2 * mu * mu * x));
    // http://docs.scipy.org/doc/scipy-0.14.0/reference/generated/scipy.stats.invgauss.html
    // invgauss.pdf(x, mu) = 1 / sqrt(2*pi*x**3) * exp(-(x-mu)**2/(2*x*mu**2))
}